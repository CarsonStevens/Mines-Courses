/*
 * Author: Carson Stevens
 * Date: April 2, 2019
 * Description: 	1)	Read in input from file
 *                  2)	Perform Sparse Matrix Vector
 *                      Multiplication on read in matrix
 */



#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <fstream>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>


using namespace std;

__global__ void spmv(const int num_rows, const int* ptr, const int* indices,
                     const float* data, const float* mult_data, float* result){

    // WORKING
    // Cache the rows of mult_data[] corresponding to this block.
    extern __shared__ float cache[];

    int block_begin = blockIdx.x * blockDim.x;
    int block_end = block_begin + blockDim.x;
    int row = block_begin + threadIdx.x;

    // Fetch and cache window of mult_data[].
    if( row < num_rows){
        cache[threadIdx.x] = mult_data[row];
    }
    __syncthreads();

    if(row < num_rows){

        int row_begin = ptr[row];
        int row_end = ptr[row+1];
        float mult_temp = 0;
        float sum = 0 ;

        for(int col = row_begin; col < row_end; ++col){
            int j = indices[col];

            // Fetch updated mult_temp from our cache when possible
            if(j >= block_begin && j < block_end) {
                mult_temp = cache[j-block_begin];
            }
            else {
                mult_temp = mult_data[j];
            }

            //Sum the dot product
            sum += data[col] * mult_temp;
        }

        //Send updated sum to the result matrix when all threads done.
        result[row] = sum;
    }

    /* WORKING: No optimization
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < num_rows) {
        float dot = 0.0;
        int row_start = ptr[row];
        int row_end = ptr[row + 1];

        // Compute sum per thread
        for (int i = row_start; i < row_end; i++) {
            dot += data[i] * mult_data[indices[i]];
        }

        result[row] = dot;
    }
    */

    /* NOT WORKING
    extern __shared__ float vals[];

    // global thread indexes
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

    // global warp index
    int warp_id = thread_id/32;

    // thread index inside warp
    int lane = thread_id % 32;

    // one warp per row
    int row = warp_id;

    if (row < num_rows){

        int row_start = ptr[row];
        int row_end = ptr[row+1];

        // Compute sum per thread
        for(int i = row_start + lane; i < row_end; i++){
            vals[threadIdx.x] += data[i] * mult_data[indices[i]];
        }

        //Synchronization for shared memory
        if(lane < 16){
            vals[threadIdx.x] += vals[threadIdx.x + 16];
        }
        if(lane < 8){
            vals[threadIdx.x] += vals[threadIdx.x + 8];
        }
        if(lane < 4){
            vals[threadIdx.x] += vals[threadIdx.x + 4];
        }
        if(lane < 2){
            vals[threadIdx.x] += vals[threadIdx.x + 2];
        }
        if(lane < 1){
            vals[threadIdx.x] += vals[threadIdx.x + 1];
        }

        // first thread writes the result
        if(lane == 0){
            result[row] += vals[threadIdx.x];
        }
    }
    */

}

int main(int argc, char* argv[]){

    srand(0);
    //Load file
    //Declare ifstream object for .txt file parsing.
    //open the file from which to read the data
    ifstream file(argv[1]);
    if (!file) {
        cerr << "Error opening input:\t" << argv[1] << endl;
        return (1);
    }

    int num_cols;
    int num_rows;
    int number_of_entries;
    file >> num_cols >> num_rows >> number_of_entries;

    // Define matrices for computation
    int column[number_of_entries];
    int row_ptr[num_rows+1];
    float data[number_of_entries];
    float mult_data[num_cols];
    float result[num_rows];

    int* dev_columns;
    int* dev_row_ptr;
    float* dev_data;
    float* dev_mult_data;
    float* dev_result;

    //Initialize the result array to 0
    for(int i = 0; i < num_rows; i++){
        result[i] = 0.0;
    }
    //Initialize the multiply vector with data
    for(int i = 0; i < num_cols; i++){
        mult_data[i] = (rand() % 100000000) / 111111111.0;
    }

    //Markers for keeping track of data
    int counter = 0;
    int ptr_counter = 0;
    int last_row = 0;
    int current_row = 0;

    while (counter < number_of_entries){

        //Read in values
        file >> current_row >> column[counter] >> data[counter];

        //-1 to change read in form to zero indexing
        column[counter]--;

        //Check to see if new entry for row_ptr
        if(current_row != last_row){
            row_ptr[current_row-1] = counter;
            ptr_counter++;
            last_row = current_row;
        }
        counter++;
    }
    row_ptr[num_rows] = number_of_entries;
    file.close();

    int size_int = sizeof(int);
    int size_float = sizeof(float);

    // Allocate memory on GPU
    hipMalloc((void **)&dev_columns, size_int*number_of_entries);
    hipMalloc((void **)&dev_row_ptr, size_int*(num_rows+1));
    hipMalloc((void **)&dev_data, size_float*number_of_entries);
    hipMalloc((void **)&dev_mult_data, size_float*num_cols);
    hipMalloc((void **)&dev_result, size_float*num_rows);

    // copy data to device
    hipMemcpy(dev_columns, column, size_int*number_of_entries, hipMemcpyHostToDevice);
    hipMemcpy(dev_row_ptr, row_ptr, size_int*(num_rows+1), hipMemcpyHostToDevice);
    hipMemcpy(dev_data, data, size_float*number_of_entries, hipMemcpyHostToDevice);
    hipMemcpy(dev_mult_data, mult_data, size_float*num_cols, hipMemcpyHostToDevice);
    hipMemcpy(dev_result, result, size_float*num_rows, hipMemcpyHostToDevice);

    // Establish thread and block size
    int minGridSize;
    int blockSize;
    int gridSize;
    //Optimization function
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, spmv, 0, number_of_entries);

    // Round up according to array size
    gridSize = (number_of_entries + blockSize - 1) / blockSize;
    // Call function
    // second blockSize for shared memory
    spmv<<<gridSize, blockSize, blockSize>>>(num_rows, dev_row_ptr, dev_columns, dev_data, dev_mult_data, dev_result);

    // copy result back
    hipMemcpy(result, dev_result, size_float*num_rows, hipMemcpyDeviceToHost);

    // free memory
    hipFree(dev_columns);
    hipFree(dev_row_ptr);
    hipFree(dev_data);
    hipFree(dev_mult_data);
    hipFree(dev_result);

    //To Print result
    cout << "[ ";
    for(int i = 0; i < num_rows; i++){
        cout << result[i] << " ";
    }
    cout << "]" << endl;

}