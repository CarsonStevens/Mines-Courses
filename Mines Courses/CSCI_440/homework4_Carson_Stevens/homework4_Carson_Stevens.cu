#include "hip/hip_runtime.h"
//
// Created by steve on 4/19/2019.
//

#include <iostream>
#include <cstdlib>
#include <random>
#include <ctime>

using namespace std;

__device__ bool lastBlock(int* counter) {
    __threadfence(); //ensure that partial result is visible by all blocks
    int last = 0;
    if (threadIdx.x == 0){
        last = atomicAdd(counter, 1);
    }
    return __syncthreads_or(last == gridDim.x-1);
}

__global__ void scan_with_addition(const int N, const int* sum_array, const int* A_gpu, int* lastBlockCounter) {

    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockDim.x;
    const int gridSize = blockSize*gridDim.x;
    int sum = 0;
    for (int i = gthIdx; i < N; i += gridSize){
        sum += sum_array[i];
    }

    __shared__ int shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (thIdx<size)
            shArr[thIdx] += shArr[thIdx+size];
        __syncthreads();
    }
    if (thIdx == 0){
        A_gpu[blockIdx.x] = shArr[0];
    }

    if (lastBlock(lastBlockCounter)) {
        shArr[thIdx] = thIdx<gridSize ? A_gpu[thIdx] : 0;
        __syncthreads();
        for (int size = blockSize/2; size>0; size/=2) { //uniform
            if (thIdx<size){
                shArr[thIdx] += shArr[thIdx+size];
            }
            __syncthreads();
        }
        if (thIdx == 0){
            A_gpu[0] = shArr[0];
        }

    }
}

int main(int argc, char* argv[]) {

    srand(time(0));
    int N = (int)argv[1];
    int sum_array[N];
    int A_cpu[N];
    int A_gpu[N];
    int *dev_sum_array[N];
    int *dev_A_gpu[N];

    // Initialize array to be summed
    for(int i = 0; i < N; i++){
        sum_array[i] = rand()%1000 + 1;
    }

    // Compute A_cpu
    A_cpu[0] = 0;
    for(int i = 1; i < N; i++){
        A_cpu[i] = sum_array[i-1] + A_cpu[i-1];
        //cout << A_cpu[i] << endl;
    }


    hipMalloc((void **)&dev_sum_array, sizeof(int)*N);
    hipMalloc((void **)&dev_A_gpu, sizeof(int)*N);

    // copy data to device
    hipMemcpy(dev_sum_array, sum_array, sizeof(int)*N, hipMemcpyHostToDevice);
    hipMemcpy(dev_A_gpu, A_gpu, sizeof(int)*N, hipMemcpyHostToDevice);


    // Establish thread and block size
    int minGridSize;
    int blockSize;
    int gridSize;

    int* dev_lastBlockCounter;
    hipMalloc((void**)&dev_lastBlockCounter, sizeof(int));
    hipMemset(dev_lastBlockCounter, 0, sizeof(int));

    //Optimization function
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, scan_with_addition, 0, N);

    // Round up according to array size
    gridSize = (N + blockSize - 1) / blockSize;

    // Call function
    // second blockSize for shared memory
    scan_with_addition<<<gridSize, blockSize, blockSize>>>(N, dev_sum_array, dev_A_gpu, dev_lastBlockCounter);
    hipDeviceSynchronize();

    // copy result back
    hipMemcpy(A_gpu, dev_A_gpu, sizeof(int)*N, hipMemcpyDeviceToHost);

    // free memory
    hipFree(dev_sum_array);
    hipFree(dev_A_gpu);

    cout << ">>>\tTESTING RESULTS BY COMPARISION\t<<<" << endl << endl;
    bool check = true;
    int break_index = 0;
    for(int i = 0; i < N; i++){
        if(A_gpu[i] != A_cpu[i]){
            check = false;
            break_index = i;
            break;
        }
    }

    if(check){
        cout << "Tested arrays are equivalent." << endl;
    }
    else{
        cout << "FAILED @ INDEX: " << break_index << endl;
    }

    return 0;
}
