//
// Created by steve on 4/19/2019.
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <random>
#include <ctime>
#include <chrono>

using namespace std;

__global__ void scan_with_addition(int* sum_array, int* A_gpu, const int N) {
    int tid = threadIdx.x;
    extern __shared__ int temp[];

    int out = 0;
    int in = 1;

    temp[tid] = (tid > 0) ? sum_array[tid-1] : 0;
    __syncthreads();

    for(int offset=1; offset < N; offset *= 2){
        out = 1 - out;
        in = 1 - out;

        if(tid >= offset){
            temp[out*N+tid] = temp[in*N+tid-offset] + temp[in*N+tid];
        }
        else{
            temp[out*N+tid] = temp[in*N+tid];
        }

        __syncthreads();
    }

    A_gpu[tid] = temp[out*N+tid];
}

__global__ void prescan(int *g_odata, int *g_idata, int n){
    extern __shared__ int temp[];// allocated on invocation


    int thid = threadIdx.x;
    int offset = 1;

    temp[2*thid] = g_idata[2*thid]; // load input into shared memory
    temp[2*thid+1] = g_idata[2*thid+1];
    // build sum in place up the tree
    for (int d = n>>1; d > 0; d >>= 1){


        __syncthreads();

        if (thid < d){
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (thid == 0) { temp[n - 1] = 0; } // clear the last element

    // traverse down tree & build scan
    for (int d = 1; d < n; d *= 2){
        offset >>= 1;
        __syncthreads();
        if (thid < d){
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            float t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();
    g_odata[2*thid] = temp[2*thid]; // write results to device memory
    g_odata[2*thid+1] = temp[2*thid+1];
}


string speedup(double baseline_duration, double duration) {
    double speedup = baseline_duration / duration;
    return to_string(speedup) + " times ";
}

int main(int argc, char *argv[]) {
    ///////////////////////////////////////////
    //SETUP
    ///////////////////////////////////////////
    srand(time(0));
    const int N = atoi(argv[1]);
    int sum_array[N];
    int A_cpu[N];
    int A_gpu[N];
    int *dev_sum_array;
    int *dev_A_gpu;


    ///////////////////////////////////////////
    // Array Initialization
    ///////////////////////////////////////////

    // Initialize array to be summed
    for (int i = 0; i < N; i++) {
        sum_array[i] = rand() % 1000 + 1;
    }

    // Compute A_cpu
    auto start = chrono::high_resolution_clock::now();
    A_cpu[0] = 0;
    for (int i = 1; i < N; i++) {
        A_cpu[i] = sum_array[i - 1] + A_cpu[i - 1];
        //cout << A_cpu[i] << endl;
    }
    auto stop = chrono::high_resolution_clock::now();
    auto baseline = stop - start;


    ///////////////////////////////////////////
    // CUDA
    ///////////////////////////////////////////

    // copy data to device
    hipMalloc((void **) &dev_sum_array, sizeof(int) * N);
    hipMalloc((void **) &dev_A_gpu, sizeof(int) * N);
    hipMemcpy(dev_sum_array, sum_array, sizeof(int)*N, hipMemcpyHostToDevice);
    hipMemcpy(dev_A_gpu, A_gpu, sizeof(int)*N, hipMemcpyHostToDevice);

    dim3  blocksize(N);
    dim3 gridsize(1);
    // Call function
    start = chrono::high_resolution_clock::now();
    //reduce<<< gridsize, blocksize >>>(dev_sum_array,dev_A_gpu);
    prescan<<< gridsize, blocksize, 2*N*sizeof(int) >>>(dev_sum_array, dev_A_gpu, N);
    hipDeviceSynchronize();
    stop = chrono::high_resolution_clock::now();
    auto real = stop - start;

    // copy result back
    hipMemcpy(A_gpu, dev_A_gpu, sizeof(int)*N, hipMemcpyDeviceToHost);

    // free memory
    hipFree(dev_sum_array);
    hipFree(dev_A_gpu);




    /////////////////////////////////////////////////
    // TESTING/VALIDITY
    /////////////////////////////////////////////////
    cout << ">>>\tTESTING RESULTS BY COMPARISION\t<<<" << endl << endl;
    bool check = true;
    int break_index = 0;
    for(int i = 0; i < N; i++){
        if(A_gpu[i] != A_cpu[i]){
            cout << "GPU:\t" << A_gpu[i] << endl << "CPU:\t" << A_cpu[i] << endl << endl;
            check = false;
            break_index = i;
            break;
        }
    }
    if(check){
        cout << "Tested arrays are equivalent." << endl;
        cout << "\tSpeed up measured at " << speedup(chrono::duration <double, milli> (baseline).count(), chrono::duration <double, milli>
                (real).count()) << "the baseline." << endl;
    }
    else{
        cout << "FAILED @ INDEX: " << break_index << endl;
    }

    return 0;
}
