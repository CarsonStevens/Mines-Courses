/*
 * Author: Carson Stevens
 * Date: February 27, 2019
 * Description: 	1)	Read in input from m1.txt into a matrix
 *                  2)	Use CUDA to return the same matrix transposed
 */



#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <fstream>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>

using namespace std;

//__global__ void transpose_matrix(int *dev_transpose, int *dev_matrix, int width, int height){
//
//    int row = threadIdx.x + blockDim.x * blockIdx.x;
//    int col = threadIdx.y + blockDim.y * blockIdx.y;
////    int width_block = blockDim.x * blockDim.y;
//
//    dev_transpose[row*height + col] = dev_matrix[col*width + row];
//
//    //Mapping for transpose
////    for (int j = 0; j < blockDim.x; j+= width){
////        transpose[x * width_block + (y + j)] = matrix[(y + j) * width_block + x];
////    }
//}
//
//
//int main( int argc, char* argv[] ) {
//
//    //Load file
//    //Declare ifstream object for .txt file parsing.
//    //open the file from which to read the data
//    ifstream file(argv[1]);
//    if (!file) {
//        cerr << "Error opening input." << endl;
//        return (1);
//    }
//
//    // for reading in values from .txt
//    int width = 0;
//    int height = 0;
//    string str;
//    int temp;
//
//    file >> width >> height;
//    // Define matrices for original and transpose
//    int matrix[width][height];
//    int transpose[height][width];
//    int *dev_transpose;
//    int *dev_matrix;
//
//    for (int i = 0; i < height; i++){
//        for (int j = 0; j < width; j++){
//            file >> str;
//            temp = atoi(str.c_str());
//            matrix[i][j] = temp;
//            //file >> matrix[i][j];
//            //cout << "(" << i << "," << j << ")\t" << matrix[i][j] << endl;
//        }
//    }
//    file.close();
//
//    //int size = sizeof(int);
//
//    // allocate memory on device
//    cudaMalloc((void **)&dev_matrix,height*width*sizeof(int));
//    cudaMalloc((void **)&dev_transpose,height*width*sizeof(int));
//
//    // copy host data to device using cudaMemcpy
//    cudaMemcpy(dev_matrix,matrix,width*height*sizeof(int),cudaMemcpyHostToDevice);
//
//    // kernel call
//    dim3 threadsPerBlock(width,height,1);
//    dim3 numBlocks((width+threadsPerBlock.x-1)/threadsPerBlock.x,
//                   (height+threadsPerBlock.y-1)/threadsPerBlock.y,1);
//
//    transpose_matrix<<<numBlocks, threadsPerBlock>>>(dev_transpose,dev_matrix,width,height);
//
//    // copy result from device to host
//    cudaMemcpy(transpose,dev_transpose,width*height*sizeof(int),cudaMemcpyDeviceToHost);
////    //Allocate CUDA space
////    cudaMalloc((void **) &dev_matrix, width * height * size);
////    cudaMalloc((void **) &dev_transpose, width * height * size);
////
////    cudaMemcpy(dev_matrix, matrix, width * height * size, cudaMemcpyHostToDevice);
////    //cudaMemcpy(dev_transpose, transpose, width * height * size, cudaMemcpyHostToDevice);
////
////    dim3 dimBlock(width, height, 1);
////    dim3 numBlock(((width+dimBlock.x-1)/dimBlock.x), ((height+dimBlock.y-1)/dimBlock.y), 1);
////
////    transpose_matrix<<<numBlock, dimBlock>>>(dev_transpose, dev_matrix, width, height);
////    cudaMemcpy(transpose, &dev_transpose, size*width*height, cudaMemcpyDeviceToHost);
//
//
//    //Print results to output
//    cout << "original" << endl << width << " " << height << endl;
//
//    for (int i = 0; i < height; i++){
//        for(int j = 0; j < width; j++){
//            cout << matrix[i][j] << " ";
//        }
//        cout << endl;
//    }
//
//    cout << "transpose" << endl;
//    for (int i = 0; i < width; i++){
//        for(int j = 0; j < height; j++){
//            cout << transpose[i][j] << " ";
//        }
//        cout << endl;
//    }
//
//    cudaFree(dev_matrix);
//    cudaFree(dev_transpose);
//
//}
__global__ void matrix_transpose(int* dev_transpose, const int* dev_matrix, int width, int height){
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    //Mapping (2d array to 1D)
    dev_transpose[col*height + row] = dev_matrix[row*width + col];
}

int main(int argc, char* argv[]){
//    ifstream file;
//    string str;
//    file.open(argv[1]); // create stream to read txt file
//
//    int col,row;
//    file >> str;
//    col = atoi(str.c_str());
//    file >> str;
//    row = atoi(str.c_str());
//
//    // host variables
//    int matrix[row][col];
//    int transpose[col][row];
//
//    // device variables
//    int *dev_matrix;
//    int *dev_transpose;
//
//    int temp;
//    // populate host matrix from text file
//    for(int i=0;i<row;i++){
//        for(int j=0;j<col;j++){
//            file >> str;
//            temp = atoi(str.c_str());
//            matrix[i][j] = temp;
//        }
//    }
//    file.close();

    //Load file
    //Declare ifstream object for .txt file parsing.
    //open the file from which to read the data
    ifstream file(argv[1]);
    if (!file) {
        cerr << "Error opening input." << endl;
        return (1);
    }

    int col,row;
    string str;
    int temp;
    file >> str;
    col = atoi(str.c_str());
    file >> str;
    row = atoi(str.c_str());


    //file >> col >> row;
    // Define matrices for original and transpose
    int matrix[row][col];
    int transpose[col][row];
    int *dev_transpose;
    int *dev_matrix;
    for(int i=0;i<row;i++){
        for(int j=0;j<col;j++){
            file >> str;
            temp = atoi(str.c_str());
            matrix[i][j] = temp;
        }
    }
    file.close();
//    for (int i = 0; i < row; i++){
//        for (int j = 0; j < col; j++){
//            file >> str;
//            temp = atoi(str.c_str());
//            matrix[i][j] = temp;
//            //file >> matrix[i][j];
//            //cout << "(" << i << "," << j << ")\t" << matrix[i][j] << endl;
//        }
//    }
//    file.close();
    // allocate memory on device
    hipMalloc((void **)&dev_matrix,row*col*sizeof(int));
    hipMalloc((void **)&dev_transpose,row*col*sizeof(int));

    // copy host data to device using cudaMemcpy
    hipMemcpy(dev_matrix,matrix,row*col*sizeof(int),hipMemcpyHostToDevice);

    // kernel call
    dim3 threadsPerBlock(col,row,1);
    dim3 numBlocks((col+threadsPerBlock.x-1)/threadsPerBlock.x,
                   (row+threadsPerBlock.y-1)/threadsPerBlock.y,1);

    matrix_transpose<<<numBlocks, threadsPerBlock>>>(dev_transpose,dev_matrix,col,row);

    // copy result from device to host
    hipMemcpy(transpose,dev_transpose,row*col*sizeof(int),hipMemcpyDeviceToHost);

    cout << "\n original"<<endl;
    for(int i=0;i<row;i++){
        cout << "\n";
        for(int j=0;j<col;j++){
            cout << matrix[i][j] << " ";
        }
    }

    // print result
    cout << "\n transposed" << endl;
    for(int i=0;i<col;i++){
        cout << "\n";
        for(int j=0;j<row;j++){
            cout << transpose[i][j] << " ";
        }
    }
    cout << endl;

    // free memory on device
    hipFree(dev_matrix);
    hipFree(dev_transpose);

    return 0;
}
